#include "hip/hip_runtime.h"
#include <glad/glad.h>

#include "glcontroller.cuh"

#include "../meta_factory/blood_cell_factory.hpp"
#include "../objects/vein_triangles.cuh"
#include "../utilities/cuda_handle_error.cuh"
#include "../utilities/cuda_vec3.cuh"
#include "../utilities/cuda_threads.hpp"

#include <functional>
#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include "cudaGL.h"
#include "cuda_gl_interop.h"



namespace graphics
{

	template<int bloodCellCount, int particlesInBloodCell, int particlesStart, int bloodCellTypeStart>
	__global__ void calculatePositionsKernel(float* devCudaPositionsBuffer, cudaVec3 positions)
	{
		int relativeId = blockIdx.x * blockDim.x + threadIdx.x;
		if (relativeId >= particlesInBloodCell * bloodCellCount)
			return;
		int id = particlesStart + relativeId;

		devCudaPositionsBuffer[6 * relativeId] = positions.x[id];
		devCudaPositionsBuffer[6 * relativeId + 1] = positions.y[id];
		devCudaPositionsBuffer[6 * relativeId + 2] = positions.z[id];
	}

	__global__ void calculateTriangleVerticesKernel(float* devVeinVBOBuffer, cudaVec3 positions, int vertexCount)
	{
		int id = blockIdx.x * blockDim.x + threadIdx.x;
		if (id >= vertexCount)
			return;

		// Insert any debug position changes here
		float3 v = positions.get(id);
		devVeinVBOBuffer[6 * id] = v.x;
		devVeinVBOBuffer[6 * id + 1] = v.y;
		devVeinVBOBuffer[6 * id + 2] = v.z;
	}

	void* mapResourceAndGetPointer(hipGraphicsResource_t resource)
	{
		// get CUDA a pointer to openGL buffer
		void* resourceBuffer = 0;
		size_t numBytes;

		HANDLE_ERROR(hipGraphicsMapResources(1, &resource, 0));
		HANDLE_ERROR(hipGraphicsResourceGetMappedPointer((void**)&resourceBuffer, &numBytes, resource));
		return resourceBuffer;
	}


	GLController::GLController(Mesh& veinMesh, std::vector<glm::vec3>& initialPositions)
	{
		veinModel.addMesh(veinMesh);
		// Register OpenGL buffer in CUDA for vein
		HANDLE_ERROR(hipGraphicsGLRegisterBuffer(&cudaVeinVBOResource, veinModel.getVboBuffer(0), hipGraphicsRegisterFlagsNone));

		using TypeList = mp_iota_c<bloodCellTypeCount>;
		std::array<unsigned int, bloodCellTypeCount> VBOs;
		mp_for_each<TypeList>([&](auto typeIndex) 
		{
			std::vector<Vertex> vertices;
			std::vector<unsigned int> indices;
			using BloodCellDefinition = mp_at_c<BloodCellList, typeIndex>;
			constexpr int verticesCount = BloodCellDefinition::particlesInCell;

			using verticeIndexList = mp_iota_c<verticesCount>;

			using VerticeList = typename BloodCellDefinition::Vertices;
			using NormalList = typename BloodCellDefinition::Normals;
			using IndiceList = typename BloodCellDefinition::Indices;

			mp_for_each<verticeIndexList>([&](auto i)
				{
					Vertex v;
					v.position = glm::vec3(
						mp_at_c<VerticeList, i>::x,
						mp_at_c<VerticeList, i>::y,
						mp_at_c<VerticeList, i>::z
					);
					v.normal = glm::vec3(
						mp_at_c<NormalList, i>::x,
						mp_at_c<NormalList, i>::y,
						mp_at_c<NormalList, i>::z
					);
					vertices.push_back(v);
				});

			using indiceIndexList = mp_iota_c<mp_size<IndiceList>::value>;
			mp_for_each<indiceIndexList>([&](auto i)
				{
					indices.push_back(mp_at_c<IndiceList, i>::value);
				});

			std::vector<glm::vec3> bloodCellInitials(BloodCellDefinition::count);
			auto initialsIterStart = initialPositions.begin() + bloodCellTypesStarts[typeIndex];
			std::copy(initialsIterStart, initialsIterStart + BloodCellDefinition::count, bloodCellInitials.begin());

			bloodCellmodel[typeIndex] = MultipleObjectModel(std::move(vertices), std::move(indices), bloodCellInitials, BloodCellDefinition::count);
			VBOs[typeIndex] = bloodCellmodel[typeIndex].getVboBuffer(0);
			// Register OpenGL buffer in CUDA for blood cell
			HANDLE_ERROR(hipGraphicsGLRegisterBuffer(&(cudaPositionsResource[typeIndex]), bloodCellmodel[typeIndex].getVboBuffer(0), hipGraphicsRegisterFlagsNone));
			HANDLE_ERROR(hipPeekAtLastError());

			// create diffuse color for blood cell type
			vec3 color;
			color.b = float(BloodCellDefinition::color & 0xFF)/255.0f;
			color.g = float((BloodCellDefinition::color >> 8) & 0xFF)/255.0f;
			color.r = float((BloodCellDefinition::color >> 16) & 0xFF)/255.0f;
			bloodCellTypeDiffuse[typeIndex] = color;
		});
		springLines.constructSprings(VBOs);

		// Create a directional light
		directionalLight = DirLight
		{
			{
				vec3(0.4f, 0.4f, 0.4f), vec3(1, 1, 1), vec3(1, 1, 1)
			},
			vec3(0, 0, -1.0f)
		};

		// Create the shaders
		if constexpr (!useLighting)
			solidColorShader = std::make_unique<SolidColorShader>();
		else
			phongForwardShader = std::make_unique<PhongForwardShader>();
		
		cylinderSolidColorShader = std::make_unique<CylinderSolidColorShader>();
		springShader = std::make_unique<SpringShader>();

		// Create streams
		for (int i = 0; i < bloodCellTypeCount; i++)
		{
			streams[i] = hipStream_t();
			HANDLE_ERROR(hipStreamCreate(&streams[i]));
		}
	}

	GLController::~GLController()
	{
		for (int i = 0; i < bloodCellTypeCount; i++)
		{
			HANDLE_ERROR(hipStreamDestroy(streams[i]));
		}
	}

	void GLController::calculatePositions(cudaVec3 positions)
	{
		using TypeList = mp_iota_c<bloodCellTypeCount>;
		mp_for_each<TypeList>([&](auto typeIndex) 
		{
			// get CUDA a pointer to openGL buffer
			// jak cos to to do odkomentowania
			float* devCudaPositionBuffer = (float*)mapResourceAndGetPointer(cudaPositionsResource[typeIndex]);
			using BloodCellDefinition = mp_at_c<BloodCellList, typeIndex>;

			constexpr int particlesStart = particlesStarts[typeIndex];
			constexpr int bloodCellTypeStart = bloodCellTypesStarts[typeIndex];

			CudaThreads threads(BloodCellDefinition::count * BloodCellDefinition::particlesInCell);
			// translate our CUDA positions into Vertex offsets
			calculatePositionsKernel<BloodCellDefinition::count, BloodCellDefinition::particlesInCell, particlesStart, bloodCellTypeStart>
				<< <threads.blocks, threads.threadsPerBlock, 0, streams[typeIndex] >> > (devCudaPositionBuffer, positions);
			HANDLE_ERROR(hipPeekAtLastError());
			HANDLE_ERROR(hipGraphicsUnmapResources(1, &cudaPositionsResource[typeIndex], 0));
			HANDLE_ERROR(hipPeekAtLastError());
		});
	}

	void GLController::calculateTriangles(VeinTriangles triangles)
	{
		// map vertices
		float* vboPtr = (float*)mapResourceAndGetPointer(cudaVeinVBOResource);
		int threadsPerBlock = triangles.vertexCount > 1024 ? 1024 : triangles.vertexCount;
		int blocks = (triangles.vertexCount + threadsPerBlock - 1) / threadsPerBlock;
		calculateTriangleVerticesKernel << <blocks, threadsPerBlock >> > (vboPtr, triangles.positions, triangles.vertexCount);
		HANDLE_ERROR(hipPeekAtLastError());
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGraphicsUnmapResources(1, &cudaVeinVBOResource, 0));
		HANDLE_ERROR(hipPeekAtLastError());
	}

	void GLController::draw(Camera& camera)
	{
		// Draw particles
		if constexpr (!useLighting) // solidcolor
		{
			solidColorShader->use();
			solidColorShader->setMatrix("model", model);
			solidColorShader->setMatrix("view", camera.getView());
			solidColorShader->setMatrix("projection", projection);

			using TypeList = mp_iota_c<bloodCellTypeCount>;
			mp_for_each<TypeList>([&](auto typeIndex)
				{
					bloodCellmodel[typeIndex].draw(solidColorShader.get());
				});
		}
		else
		{
			phongForwardShader->use();
			phongForwardShader->setMatrix("model", model);
			phongForwardShader->setMatrix("view", camera.getView());
			phongForwardShader->setMatrix("projection", projection);

			phongForwardShader->setVector("viewPos", camera.getPosition());
			phongForwardShader->setFloat("Specular", particleSpecular);
			phongForwardShader->setFloat("Shininess", 32);

			phongForwardShader->setLighting(directionalLight);

			mp_for_each<mp_iota_c<bloodCellTypeCount>>([&](auto typeIndex)
				{
					phongForwardShader->setVector("Diffuse", bloodCellTypeDiffuse[typeIndex]);
					bloodCellmodel[typeIndex].draw(phongForwardShader.get());
				});
		}

		if (BLOOD_CELL_SPRINGS_RENDER)
		{
			// Draw lines
			springShader->use();
			springShader->setMatrix("projection_view_model", projection * camera.getView());
			springLines.draw(springShader.get());
		}

		// Draw vein
		cylinderSolidColorShader->use();
		cylinderSolidColorShader->setMatrix("view", camera.getView());
		cylinderSolidColorShader->setMatrix("projection", projection);

		glDisable(GL_CULL_FACE);
		//glCullFace(GL_FRONT);
		veinModel.draw(cylinderSolidColorShader.get());
		//glCullFace(GL_BACK);
		glEnable(GL_CULL_FACE);
	}
}