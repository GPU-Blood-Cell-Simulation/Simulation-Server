#include "hip/hip_runtime.h"
#include <glad/glad.h>

#include "glcontroller.cuh"

#include "../meta_factory/blood_cell_factory.hpp"
#include "../objects/vein_triangles.cuh"
#include "../utilities/cuda_handle_error.cuh"
#include "../utilities/cuda_vec3.cuh"
#include "../utilities/cuda_threads.hpp"

#include <functional>
#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include "cudaGL.h"
#include "cuda_gl_interop.h"



namespace graphics
{
	// __global__ void calculateOffsetsKernel(float* devCudaOffsetBuffer, cudaVec3 positions)
	// {
	// 	int id = blockIdx.x * blockDim.x + threadIdx.x;
	// 	if (id >= particleCount)
	// 		return;
		
	// 	//printf("[%d] x=%.5f, y=%.5f, z=%.5f\n", id, positions.x[id], positions.y[id], positions.z[id]);
	// 	devCudaOffsetBuffer[3 * id] = positions.x[id];
	// 	devCudaOffsetBuffer[3 * id + 1] = positions.y[id];
	// 	devCudaOffsetBuffer[3 * id + 2] = positions.z[id];

	// }

	template<int bloodCellCount, int particlesInBloodCell, int particlesStart, int bloodCellTypeStart>
	__global__ void calculatePositionsKernel(float* devCudaPositionsVertices, float* devCudaPositionsOffsets, cudaVec3 positions)
	{
		int relativeId = blockIdx.x * blockDim.x + threadIdx.x;
		if (relativeId >= particlesInBloodCell * bloodCellCount)
			return;
		int id = particlesStart + relativeId;

		devCudaPositionsVertices[6 * relativeId] = positions.x[id];
		devCudaPositionsVertices[6 * relativeId + 1] = positions.y[id];
		devCudaPositionsVertices[6 * relativeId + 2] = positions.z[id];
		devCudaPositionsOffsets[3 * id] = positions.x[id];
		devCudaPositionsOffsets[3 * id + 1] = positions.y[id];
		devCudaPositionsOffsets[3 * id + 2] = positions.z[id];
	}

	__global__ void calculateTriangleVerticesKernel(float* devVeinVBOBuffer, cudaVec3 positions, int vertexCount)
	{
		int id = blockIdx.x * blockDim.x + threadIdx.x;
		if (id >= vertexCount)
			return;

		// Insert any debug position changes here
		float3 v = positions.get(id);
		devVeinVBOBuffer[6 * id] = v.x;
		devVeinVBOBuffer[6 * id + 1] = v.y;
		devVeinVBOBuffer[6 * id + 2] = v.z;
	}

	void* mapResourceAndGetPointer(hipGraphicsResource_t resource)
	{
		// get CUDA a pointer to openGL buffer
		void* resourceBuffer = 0;
		size_t numBytes;

		HANDLE_ERROR(hipGraphicsMapResources(1, &resource, 0));
		HANDLE_ERROR(hipGraphicsResourceGetMappedPointer((void**)&resourceBuffer, &numBytes, resource));
		return resourceBuffer;
	}


	GLController::GLController(SingleObjectMesh& veinMesh, InstancedObjectMesh& sphereMesh, sim::SimulationController& simulationController): veinModel(veinMesh), cellSphereModel(sphereMesh, particleCount)
	{
		// Register OpenGL buffer in CUDA for vein
		HANDLE_ERROR(hipGraphicsGLRegisterBuffer(&cudaVeinVBOResource, veinModel.getVboBuffer(0), hipGraphicsRegisterFlagsNone));
		HANDLE_ERROR(hipGraphicsGLRegisterBuffer(&cudaOffsetResource, cellSphereModel.getCudaOffsetBuffer(), hipGraphicsRegisterFlagsNone));

		using TypeList = mp_iota_c<bloodCellTypeCount>;
		std::array<unsigned int, bloodCellTypeCount> VBOs;
		std::copy(simulationController.smallestRadiusInType.begin(), simulationController.smallestRadiusInType.end(), cellSphereRadius.begin());
		minimalRadius = *std::min_element(cellSphereRadius.begin(), cellSphereRadius.end());
		mp_for_each<TypeList>([&](auto typeIndex)
			{
				std::vector<Vertex> vertices;
				std::vector<unsigned int> indices;
				using BloodCellDefinition = mp_at_c<BloodCellList, typeIndex>;
				constexpr int verticesCount = BloodCellDefinition::particlesInCell;

				using verticeIndexList = mp_iota_c<verticesCount>;

				using VerticeList = typename BloodCellDefinition::Vertices;
				using NormalList = typename BloodCellDefinition::Normals;
				using IndiceList = typename BloodCellDefinition::Indices;

				mp_for_each<verticeIndexList>([&](auto i)
					{
						Vertex v;
						v.position = glm::vec3(
							mp_at_c<VerticeList, i>::x,
							mp_at_c<VerticeList, i>::y,
							mp_at_c<VerticeList, i>::z
						);
						v.normal = glm::vec3(
							mp_at_c<NormalList, i>::x,
							mp_at_c<NormalList, i>::y,
							mp_at_c<NormalList, i>::z
						);
						vertices.push_back(v);
					});

				using indiceIndexList = mp_iota_c<mp_size<IndiceList>::value>;
				mp_for_each<indiceIndexList>([&](auto i)
					{
						indices.push_back(mp_at_c<IndiceList, i>::value);
					});

				std::vector<glm::vec3> bloodCellInitials(BloodCellDefinition::count);
				auto initialsIterStart = simulationController.initialCellPositions.begin() + bloodCellTypesStarts[typeIndex];
				std::copy(initialsIterStart, initialsIterStart + BloodCellDefinition::count, bloodCellInitials.begin());

				bloodCellmodel[typeIndex] = MultipleObjectModel(std::move(vertices), std::move(indices), bloodCellInitials, BloodCellDefinition::count);
				VBOs[typeIndex] = bloodCellmodel[typeIndex].getVboBuffer(0);
				// Register OpenGL buffer in CUDA for blood cell
				HANDLE_ERROR(hipGraphicsGLRegisterBuffer(&(cudaPositionsResource[typeIndex]), bloodCellmodel[typeIndex].getVboBuffer(0), hipGraphicsRegisterFlagsNone));
				HANDLE_ERROR(hipPeekAtLastError());

				// create diffuse color for blood cell type
				vec3 color;
				color.b = float(BloodCellDefinition::color & 0xFF) / 255.0f;
				color.g = float((BloodCellDefinition::color >> 8) & 0xFF) / 255.0f;
				color.r = float((BloodCellDefinition::color >> 16) & 0xFF) / 255.0f;
				bloodCellTypeDiffuse[typeIndex] = color;
			});
		springLines.constructSprings(VBOs);

		// Create a directional light
		directionalLight = DirLight
		{
			{
				vec3(0.4f, 0.4f, 0.4f), vec3(1, 1, 1), vec3(1, 1, 1)
			},
			vec3(0, 0, -1.0f)
		};

		// Create the shaders
		if constexpr (!useLighting)
		{
			solidColorShader = std::make_unique<SolidColorShader>();
			solidColorSphereShader = std::make_unique<SpheresSolidColorShader>();
		}
		else
		{
			phongForwardShader = std::make_unique<PhongForwardShader>();
			phongForwardSphereShader = std::make_unique<SpheresPhongForwardShader>();
		}

		veinSolidColorShader = std::make_unique<VeinSolidColorShader>();
		springShader = std::make_unique<SpringShader>();

		// Create streams
		for (int i = 0; i < bloodCellTypeCount; i++)
		{
			streams[i] = hipStream_t();
			HANDLE_ERROR(hipStreamCreate(&streams[i]));
		}
	}

	GLController::~GLController()
	{
		for (int i = 0; i < bloodCellTypeCount; i++)
		{
			HANDLE_ERROR(hipStreamDestroy(streams[i]));
		}
	}

	void GLController::calculatePositions(cudaVec3 positions)
	{			
		float* devCudaPositionOffsets = (float*)mapResourceAndGetPointer(cudaOffsetResource);
		using TypeList = mp_iota_c<bloodCellTypeCount>;
		mp_for_each<TypeList>([&](auto typeIndex)
			{
				// get CUDA a pointer to openGL buffer
				// jak cos to to do odkomentowania
				float* devCudaPositionVertices = (float*)mapResourceAndGetPointer(cudaPositionsResource[typeIndex]);
				using BloodCellDefinition = mp_at_c<BloodCellList, typeIndex>;
					constexpr int particlesStart = particleStarts[typeIndex];
				constexpr int bloodCellTypeStart = bloodCellTypesStarts[typeIndex];
					CudaThreads threads(BloodCellDefinition::count * BloodCellDefinition::particlesInCell);
				// translate our CUDA positions into Vertex offsets
				calculatePositionsKernel<BloodCellDefinition::count, BloodCellDefinition::particlesInCell, particlesStart, bloodCellTypeStart>
					<< <threads.blocks, threads.threadsPerBlock, 0, streams[typeIndex] >> > (devCudaPositionVertices, devCudaPositionOffsets, positions);
				HANDLE_ERROR(hipPeekAtLastError());
				HANDLE_ERROR(hipGraphicsUnmapResources(1, &cudaPositionsResource[typeIndex], 0));
				HANDLE_ERROR(hipPeekAtLastError());
			});
			HANDLE_ERROR(hipGraphicsUnmapResources(1, &cudaOffsetResource, 0));
	}

	void GLController::calculateTriangles(VeinTriangles triangles)
	{
		// map vertices
		float* vboPtr = (float*)mapResourceAndGetPointer(cudaVeinVBOResource);
		int threadsPerBlock = triangles.vertexCount > 1024 ? 1024 : triangles.vertexCount;
		int blocks = (triangles.vertexCount + threadsPerBlock - 1) / threadsPerBlock;
		calculateTriangleVerticesKernel << <blocks, threadsPerBlock >> > (vboPtr, triangles.positions, triangles.vertexCount);
		HANDLE_ERROR(hipPeekAtLastError());
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGraphicsUnmapResources(1, &cudaVeinVBOResource, 0));
		HANDLE_ERROR(hipPeekAtLastError());
	}

	void GLController::draw(Camera& camera)
	{
		// Draw particles
		if constexpr (!useLighting) // solidcolor
		{
			if (BLOOD_CELL_SPHERE_RENDER)
			{
				solidColorSphereShader->use();
				solidColorSphereShader->setMatrix("model", model);
				solidColorSphereShader->setMatrix("view", camera.getView());
				solidColorSphereShader->setMatrix("projection", projection);
				solidColorSphereShader->setFloat("sphereRadius", minimalRadius*float(boundingSpheresCoeff)/2);
				cellSphereModel.draw(solidColorSphereShader.get());
			}
			else
			{
				solidColorShader->use();
				solidColorShader->setMatrix("model", model);
				solidColorShader->setMatrix("view", camera.getView());
				solidColorShader->setMatrix("projection", projection);
				using TypeList = mp_iota_c<bloodCellTypeCount>;
				mp_for_each<TypeList>([&](auto typeIndex)
					{
						solidColorShader->setVector("Diffuse", bloodCellTypeDiffuse[typeIndex]);
						bloodCellmodel[typeIndex].draw(solidColorShader.get());
					});
			}
		}
		else
		{
			if (BLOOD_CELL_SPHERE_RENDER)
			{
				phongForwardSphereShader->use();
				phongForwardSphereShader->setMatrix("model", model);
				phongForwardSphereShader->setMatrix("view", camera.getView());
				phongForwardSphereShader->setMatrix("projection", projection);
				phongForwardSphereShader->setVector("viewPos", camera.getPosition());
				phongForwardSphereShader->setFloat("Specular", particleSpecular);
				phongForwardSphereShader->setFloat("Shininess", 32);
				phongForwardSphereShader->setLighting(directionalLight);
				phongForwardSphereShader->setFloat("sphereRadius", minimalRadius*float(boundingSpheresCoeff)/2);
				phongForwardSphereShader->setVector("Diffuse", bloodCellTypeDiffuse[0]); /// TODO
				cellSphereModel.draw(phongForwardSphereShader.get());
			}
			else
			{
				phongForwardShader->use();
				phongForwardShader->setMatrix("model", model);
				phongForwardShader->setMatrix("view", camera.getView());
				phongForwardShader->setMatrix("projection", projection);
				phongForwardShader->setVector("viewPos", camera.getPosition());
				phongForwardShader->setFloat("Specular", particleSpecular);
				phongForwardShader->setFloat("Shininess", 32);
				phongForwardShader->setLighting(directionalLight);
				using TypeList = mp_iota_c<bloodCellTypeCount>;
				mp_for_each<TypeList>([&](auto typeIndex)
					{
						phongForwardShader->setVector("Diffuse", bloodCellTypeDiffuse[typeIndex]);
						bloodCellmodel[typeIndex].draw(phongForwardShader.get());
					});
			}
		}

		if (BLOOD_CELL_SPRINGS_RENDER)
		{
			// Draw lines
			springShader->use();
			springShader->setMatrix("projection_view_model", projection * camera.getView());
			springLines.draw(springShader.get());
		}


		// Draw vein
		veinSolidColorShader->use();
		veinSolidColorShader->setMatrix("view", camera.getView());
		veinSolidColorShader->setMatrix("projection", projection);

		glDisable(GL_CULL_FACE);
		//glCullFace(GL_FRONT);
		veinModel.draw(veinSolidColorShader.get());
		//glCullFace(GL_BACK);
		glEnable(GL_CULL_FACE);
	}
}