#include "hip/hip_runtime.h"
#include "vein_end.cuh"

#include "../simulation/physics.cuh"
#include "../config/simulation.hpp"
#include "../meta_factory/blood_cell_factory.hpp"
#include "../meta_factory/vein_factory.hpp"
#include "../utilities/math.cuh"

#include "hip/hip_runtime.h"


constexpr float upperBoundTreshold = maxY - gridYMargin / 2;
constexpr float lowerBoundTreshold = minY + gridYMargin / 2;
constexpr float rightBoundTreshold = maxX - gridXZMargin / 2;
constexpr float leftBoundTreshold = minX + gridXZMargin / 2;
constexpr float frontBoundTreshold = maxZ - gridXZMargin / 2;
constexpr float backBoundTreshold = minZ + gridXZMargin / 2;
constexpr float targetTeleportHeight = minSpawnY;

enum SynchronizationType { warpSync, blockSync };


constexpr SynchronizationType SelectSynchronizationType(int bloodCellsCnt, int particlesInBloodCell)
{
	if (bloodCellsCnt * particlesInBloodCell <= CudaThreads::threadsInWarp ||
		CudaThreads::threadsInWarp % particlesInBloodCell == 0)
		return warpSync;

	return blockSync;
}

constexpr __host__ __device__ int CalculateThreadsPerBlock(SynchronizationType syncType, int bloodCellsCnt, int particlesInBloodCell)
{
	switch (syncType)
	{
	case warpSync:
		if (bloodCellsCnt * particlesInBloodCell < CudaThreads::threadsInWarp)
			return bloodCellsCnt * particlesInBloodCell;

		// Max number of full warps
		return (CudaThreads::maxThreadsInBlock / CudaThreads::threadsInWarp) * CudaThreads::threadsInWarp;

	case blockSync:
		// Max mulitple of number of particles in blood cell
		return (CudaThreads::maxThreadsInBlock / particlesInBloodCell) * particlesInBloodCell;

	default:
		return -1;
	}
}

constexpr int CalculateBlocksCount(SynchronizationType syncType, int particleCount, int particlesInBloodCell)
{
	return constCeil(static_cast<float>(particleCount) / CalculateThreadsPerBlock(syncType, particleCount, particlesInBloodCell));
}

template <int bloodCellsCount, int particlesInBloodCell, int particlesStart, int bloodCellmodelStart>
__global__ void handleVeinEndsBlockSync(BloodCells bloodCells, hiprandState* states, cudaVec3 bloodCellModels)
{ 
	__shared__ bool belowVein[CalculateThreadsPerBlock(blockSync, bloodCellsCount, particlesInBloodCell)];

	int indexInType = blockDim.x * blockIdx.x + threadIdx.x;

	if (indexInType >= bloodCellsCount * particlesInBloodCell)
		return;

	int realIndex = particlesStart + indexInType;
	float posX = bloodCells.particles.positions[0].x[realIndex];
	float posY = bloodCells.particles.positions[0].y[realIndex]; 
	float posZ = bloodCells.particles.positions[0].z[realIndex];

	if (posY >= upperBoundTreshold) {
		// Bounce particle off upper bound
		bloodCells.particles.velocities[0].y[realIndex] -= 5;
	}

	// Check if teleportiation should occur
	bool teleport = false;
	mp_for_each<mp_iota_c<veinEndingCenterCount>>([&](auto i)
		{
			static constexpr float endingX = mp_at_c<VeinEndingCenters, i>::x;
			static constexpr float endingY = mp_at_c<VeinEndingCenters, i>::y;
			static constexpr float endingZ = mp_at_c<VeinEndingCenters, i>::z;
			static constexpr float radius = mp_at_c<VeinEndingRadii, i>::value;
			teleport = teleport || length_squared(float3{posX - endingX, posY - endingY, posZ - endingZ}) <= radius * radius;
		});
	

	// Check additional constraints to make sure particles don't leave the grid
	teleport = teleport || posY <= lowerBoundTreshold || posX <= leftBoundTreshold || posX >= rightBoundTreshold || posZ <= backBoundTreshold || posZ >= frontBoundTreshold;
	belowVein[threadIdx.x] = teleport;

	__syncthreads();

	int particleInCellIndex = realIndex % particlesInBloodCell;
	int numberOfParticlesInThread = threadIdx.x / particlesInBloodCell * particlesInBloodCell;

	// Algorithm goes through all neighbours and checks if any of them is low enought to be teleported
#pragma unroll
	for (int i = 1; i < particlesInBloodCell; i++)
	{
		teleport |= belowVein[((particleInCellIndex + i) % particlesInBloodCell) + numberOfParticlesInThread];
	}

	if (teleport)
	{
		bloodCells.particles.positions[0].x[realIndex] = (hiprand_uniform(&states[realIndex/particlesInBloodCell]) - 0.5f) * 1.2f * cylinderRadius + bloodCellModels.x[bloodCellmodelStart + indexInType % particlesInBloodCell] - bloodCellModels.x[bloodCellmodelStart];
		bloodCells.particles.positions[0].y[realIndex] = targetTeleportHeight + bloodCellModels.y[bloodCellmodelStart + indexInType % particlesInBloodCell] - bloodCellModels.y[bloodCellmodelStart];
		bloodCells.particles.positions[0].z[realIndex] = (hiprand_uniform(&states[realIndex/particlesInBloodCell]) - 0.5f) * 1.2f * cylinderRadius + bloodCellModels.z[bloodCellmodelStart + indexInType % particlesInBloodCell] - bloodCellModels.z[bloodCellmodelStart];
		bloodCells.particles.velocities[0].set(realIndex, make_float3(initVelocityX, initVelocityY, initVelocityZ));
	}
}


template <int bloodCellsCount, int particlesInBloodCell, int particlesStart, int bloodCellmodelStart>
__global__ void handleVeinEndsWarpSync(BloodCells bloodCells, hiprandState* states, cudaVec3 bloodCellModels)
{
	int indexInType = blockDim.x * blockIdx.x + threadIdx.x;

	if (indexInType >= bloodCellsCount * particlesInBloodCell)
		return;

	int threadInWarpID = threadIdx.x % CudaThreads::threadsInWarp;

	int realIndex = particlesStart + indexInType;
	float posX = bloodCells.particles.positions[0].x[realIndex];
	float posY = bloodCells.particles.positions[0].y[realIndex]; 
	float posZ = bloodCells.particles.positions[0].z[realIndex]; 

	if (posY >= upperBoundTreshold) {
		// Bounce particle off upper bound
		bloodCells.particles.velocities[0].y[realIndex] -= 5;
	}

	static constexpr int initSyncBitMask = (particlesInBloodCell == 32) ? 0xffffffff : (1 << (particlesInBloodCell)) - 1;
	int syncBitMask = initSyncBitMask << static_cast<int>(std::floor(static_cast<float>(threadInWarpID) / particlesInBloodCell)) * particlesInBloodCell;

	// Bit mask of particles, which are below treshold
	int particlesBelowTreshold = __any_sync(syncBitMask, posY <= lowerBoundTreshold || posX <= leftBoundTreshold || posX >= rightBoundTreshold || posZ <= backBoundTreshold || posZ >= frontBoundTreshold);

	if (particlesBelowTreshold != 0) {
		bloodCells.particles.positions[0].x[realIndex] = (hiprand_uniform(&states[realIndex/particlesInBloodCell]) - 0.5f) * 1.2f * cylinderRadius + bloodCellModels.x[bloodCellmodelStart + indexInType % particlesInBloodCell] - bloodCellModels.x[bloodCellmodelStart];
		bloodCells.particles.positions[0].y[realIndex] = targetTeleportHeight + bloodCellModels.y[bloodCellmodelStart + indexInType % particlesInBloodCell] - bloodCellModels.y[bloodCellmodelStart];
		bloodCells.particles.positions[0].z[realIndex] = (hiprand_uniform(&states[realIndex/particlesInBloodCell]) - 0.5f) * 1.2f * cylinderRadius + bloodCellModels.z[bloodCellmodelStart + indexInType % particlesInBloodCell] - bloodCellModels.z[bloodCellmodelStart];
		bloodCells.particles.velocities[0].set(realIndex, make_float3(initVelocityX, initVelocityY, initVelocityZ));
	}
}


void HandleVeinEnd(BloodCells& cells, hiprandState* devStates, const std::array<hipStream_t, bloodCellTypeCount>& streams, cudaVec3& bloodCellModels)
{
	using IndexList = mp_iota_c<bloodCellTypeCount>;
	mp_for_each<IndexList>([&](auto i)
		{
			using BloodCellDefinition = mp_at_c<BloodCellList, i>;
			constexpr int particlesStart = particleStarts[i];

			constexpr SynchronizationType syncType = SelectSynchronizationType(
				BloodCellDefinition::count,
				BloodCellDefinition::particlesInCell
			);

			constexpr int threadsPerBlock = CalculateThreadsPerBlock(
				syncType,
				BloodCellDefinition::count,
				BloodCellDefinition::particlesInCell
			);

			constexpr int blocksCnt = CalculateBlocksCount(
				syncType,
				BloodCellDefinition::count * BloodCellDefinition::particlesInCell,
				BloodCellDefinition::particlesInCell
			);

			if constexpr (syncType == warpSync)
				handleVeinEndsWarpSync<BloodCellDefinition::count, BloodCellDefinition::particlesInCell, particlesStart, bloodCellModelStarts[i]>
				<< <blocksCnt, threadsPerBlock, 0, streams[i] >> > (cells, devStates, bloodCellModels);
			else if constexpr (syncType == blockSync)
				handleVeinEndsBlockSync<BloodCellDefinition::count, BloodCellDefinition::particlesInCell, particlesStart, bloodCellModelStarts[i]>
				<< <blocksCnt, threadsPerBlock, 0, streams[i] >> > (cells, devStates, bloodCellModels);
		});
}
