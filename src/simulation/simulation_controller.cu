#include "hip/hip_runtime.h"
#include "simulation_controller.cuh"

#include "../meta_factory/blood_cell_factory.hpp"
#include "../meta_factory/vein_factory.hpp"
#include "../objects/particles.cuh"
#include "particle_collisions.cuh"
#include "../utilities/cuda_handle_error.cuh"
#include "vein_collisions.cuh"
#include "vein_end.cuh"

#include <cmath>
#include <ctime>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>

namespace sim
{
	__global__ void setupCurandStatesKernel(hiprandState* states, unsigned long seed);

	template<int bloodCellCount, int particlesInBloodCell, int particlesStart, int bloodCellStart, int bloodCellmodelStar>
	__global__ void setBloodCellsPositionsFromRandom(Particles particles, cudaVec3 bloodCellModelPosition, cudaVec3 initialPositions);

	template<int totalBloodCellCount>
	__global__ void generateRandomPositonskernel(hiprandState* states, glm::vec3 cylinderBaseCenter, cudaVec3 initialPositions);

	SimulationController::SimulationController(BloodCells& bloodCells, VeinTriangles& triangles, Grid particleGrid, Grid triangleGrid) :
		bloodCells(bloodCells), triangles(triangles), particleGrid(particleGrid), triangleGrid(triangleGrid),
		bloodCellsThreads(particleCount),
		veinVerticesThreads(triangles.vertexCount),
		veinTrianglesThreads(triangles.triangleCount)
	{
		// Create streams
		for (int i = 0; i < bloodCellTypeCount; i++)
		{
			streams[i] = hipStream_t();
			HANDLE_ERROR(hipStreamCreate(&streams[i]));
		}

		// Generate random particle positions
		generateRandomPositions();
	}

	sim::SimulationController::~SimulationController()
	{
		for (int i = 0; i < bloodCellTypeCount; i++)
		{
			HANDLE_ERROR(hipStreamDestroy(streams[i]));
		}
	}

	// Generate initial positions and velocities of particles
	void SimulationController::generateRandomPositions()
	{
		// Set up random seeds
		hiprandState* devStates;
		HANDLE_ERROR(hipMalloc(&devStates, particleCount * sizeof(hiprandState)));
		srand(static_cast<unsigned int>(time(0)));
		int seed = rand();

		setupCurandStatesKernel << <bloodCellsThreads.blocks, bloodCellsThreads.threadsPerBlock >> > (devStates, seed);
		HANDLE_ERROR(hipDeviceSynchronize());

		std::vector<cudaVec3> models;
		cudaVec3 initialPositions(bloodCellCount);

		// Generate random positions and velocity vectors
		generateRandomPositonskernel<bloodCellCount> << <  bloodCellsThreads.blocks, bloodCellsThreads.threadsPerBlock >> > (devStates, cylinderBaseCenter, initialPositions);
		HANDLE_ERROR(hipDeviceSynchronize());

		float* xpos = new float[bloodCellCount];
		float* ypos = new float[bloodCellCount];
		float* zpos = new float[bloodCellCount];

		HANDLE_ERROR(hipMemcpy(xpos, initialPositions.x, bloodCellCount * sizeof(float), hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(ypos, initialPositions.y, bloodCellCount * sizeof(float), hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(zpos, initialPositions.z, bloodCellCount * sizeof(float), hipMemcpyDeviceToHost));

		for (int i = 0; i < bloodCellCount; ++i)
			initialCellPositions.push_back(glm::vec3(xpos[i], ypos[i], zpos[i]));

		delete[] xpos;
		delete[] ypos;
		delete[] zpos;

		cudaVec3 bloodCellModels = cudaVec3(particleDistinctCellsCount);
		std::array<std::array<float, particleDistinctCellsCount>, 3> hostModels;

		using IndexList = mp_iota_c<bloodCellTypeCount>;
		mp_for_each<IndexList>([&](auto i)
			{
				using BloodCellDefinition = mp_at_c<BloodCellList, i>;
				constexpr int modelSize = BloodCellDefinition::particlesInCell;
				int modelStart = bloodCellModelStarts[i];
				using verticeIndexList = mp_iota_c<modelSize>;
				using VerticeList = typename BloodCellDefinition::Vertices;

				mp_for_each<verticeIndexList>([&](auto j)
					{
						hostModels[0][modelStart + j] = mp_at_c<VerticeList, j>::x;
						hostModels[1][modelStart + j] = mp_at_c<VerticeList, j>::y;
						hostModels[2][modelStart + j] = mp_at_c<VerticeList, j>::z;
					});
			});
		HANDLE_ERROR(hipMemcpy(bloodCellModels.x, hostModels[0].data(), particleDistinctCellsCount * sizeof(float), hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(bloodCellModels.y, hostModels[1].data(), particleDistinctCellsCount * sizeof(float), hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(bloodCellModels.z, hostModels[2].data(), particleDistinctCellsCount * sizeof(float), hipMemcpyHostToDevice));

		mp_for_each<IndexList>([&](auto i)
			{
				using BloodCellDefinition = mp_at_c<BloodCellList, i>;
				constexpr int particlesStart = particlesStarts[i];
				constexpr int bloodCellTypeStart = bloodCellTypesStarts[i];
				constexpr int bloodCellModelSizesStarts = bloodCellModelStarts[i];

				CudaThreads threads(BloodCellDefinition::count * BloodCellDefinition::particlesInCell);
				setBloodCellsPositionsFromRandom<BloodCellDefinition::count, BloodCellDefinition::particlesInCell, particlesStart, bloodCellTypeStart, bloodCellModelSizesStarts>
					<< <threads.blocks, threads.threadsPerBlock, 0, streams[i] >> > (bloodCells.particles, bloodCellModels, initialPositions);
			});
		HANDLE_ERROR(hipDeviceSynchronize());
		HANDLE_ERROR(hipFree(devStates));
	}

	__global__ void setupCurandStatesKernel(hiprandState* states, unsigned long seed)
	{
		int id = blockIdx.x * blockDim.x + threadIdx.x;
		if (id >= particleCount)
			return;
		hiprand_init(seed, id, 0, &states[id]);
	}

	// generate initial positions for blood cells
	template<int totalBloodCellCount>
	__global__ void generateRandomPositonskernel(hiprandState* states, glm::vec3 cylinderBaseCenter, cudaVec3 initialPositions)
	{
		int id = blockIdx.x * blockDim.x + threadIdx.x;
		if (id >= totalBloodCellCount)
			return;
		initialPositions.x[id] = cylinderBaseCenter.x - cylinderRadius * 0.5f + hiprand_uniform(&states[id]) * cylinderRadius;
		initialPositions.y[id] = cylinderBaseCenter.y - cylinderRadius * 0.5f + hiprand_uniform(&states[id]) * 3*cylinderRadius + cylinderHeight / 2;
		initialPositions.z[id] = cylinderBaseCenter.z - cylinderRadius * 0.5f + hiprand_uniform(&states[id]) * cylinderRadius;
	}

	// Generate random positions and velocities at the beginning
	template<int bloodCellCount, int particlesInBloodCell, int particlesStart, int bloodCellTypeStart, int bloodCellmodelStart>
	__global__ void setBloodCellsPositionsFromRandom(Particles particles, cudaVec3 bloodCellModelPosition, cudaVec3 initialPositions)
	{
		int relativeId = blockIdx.x * blockDim.x + threadIdx.x;
		if (relativeId >= particlesInBloodCell * bloodCellCount)
			return;
		int id = particlesStart + relativeId;

		particles.positions.x[id] = initialPositions.x[bloodCellTypeStart + relativeId / particlesInBloodCell] + bloodCellModelPosition.x[bloodCellmodelStart + relativeId % particlesInBloodCell];
		particles.positions.y[id] = initialPositions.y[bloodCellTypeStart + relativeId / particlesInBloodCell] + bloodCellModelPosition.y[bloodCellmodelStart + relativeId % particlesInBloodCell];
		particles.positions.z[id] = initialPositions.z[bloodCellTypeStart + relativeId / particlesInBloodCell] + bloodCellModelPosition.z[bloodCellmodelStart + relativeId % particlesInBloodCell];

		particles.velocities.x[id] = initVelocityX;
		particles.velocities.y[id] = initVelocityY;
		particles.velocities.z[id] = initVelocityZ;

		particles.forces.x[id] = 0;
		particles.forces.y[id] = 0;
		particles.forces.z[id] = 0;
	}

	// Main simulation function, called every frame
	void SimulationController::calculateNextFrame()
	{
		std::visit([&](auto&& g1, auto&& g2)
			{
				// 1. Calculate grids
				// TODO: possible optimization - these grisds can be calculated simultaneously
				g1->calculateGrid(bloodCells.particles, particleCount);
				g2->calculateGrid(triangles.centers.x, triangles.centers.y, triangles.centers.z, triangles.triangleCount);

				// 2. Detect particle collisions
				calculateParticleCollisions << < bloodCellsThreads.blocks, bloodCellsThreads.threadsPerBlock >> > (bloodCells, *g1);
				HANDLE_ERROR(hipPeekAtLastError());

				// 3. Propagate particle forces into neighbors

				bloodCells.gatherForcesFromNeighbors(streams);
				HANDLE_ERROR(hipPeekAtLastError());

				// 4. Detect vein collisions and propagate forces -> velocities, velocities -> positions for particles

				detectVeinCollisionsAndPropagateParticles << < bloodCellsThreads.blocks, bloodCellsThreads.threadsPerBlock >> > (bloodCells, triangles, *g2);
				HANDLE_ERROR(hipPeekAtLastError());

				// 5. Propagate triangle forces into neighbors

				triangles.gatherForcesFromNeighbors(veinVerticesThreads.blocks, veinVerticesThreads.threadsPerBlock);
				HANDLE_ERROR(hipPeekAtLastError());

				// 6. Propagate forces -> velocities, velocities -> positions for vein triangles
				triangles.propagateForcesIntoPositions(veinVerticesThreads.blocks, veinVerticesThreads.threadsPerBlock);
				HANDLE_ERROR(hipPeekAtLastError());

				// 7. Recalculate triangles centers
				triangles.calculateCenters(veinTrianglesThreads.blocks, veinTrianglesThreads.threadsPerBlock);
				HANDLE_ERROR(hipPeekAtLastError());

				if constexpr (useBloodFlow)
				{
					HandleVeinEnd(bloodCells, streams);
					HANDLE_ERROR(hipPeekAtLastError());
				}

			}, particleGrid, triangleGrid);
	}
}