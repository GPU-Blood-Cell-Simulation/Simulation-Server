#include "hip/hip_runtime.h"
#include "blood_cells.cuh"

#include "../simulation/physics.cuh"
#include "../meta_factory/blood_cell_factory.hpp"
#include "../utilities/cuda_handle_error.cuh"
#include "../utilities/math.cuh"
#include "../utilities/cuda_threads.hpp"

#include <vector>

#include "hip/hip_runtime.h"

constexpr float NO_SPRING = 0;

BloodCells::BloodCells()
{
	HANDLE_ERROR(hipMalloc(&dev_springGraph, sizeof(float) * totalGraphSize));
	HANDLE_ERROR(hipMemcpy(dev_springGraph, springGraph.data(), sizeof(float) * totalGraphSize, hipMemcpyHostToDevice));
}

BloodCells::BloodCells(const BloodCells& other) : isCopy(true), particles(other.particles), dev_springGraph(other.dev_springGraph) {}

BloodCells::~BloodCells()
{
	if (!isCopy)
	{
		HANDLE_ERROR(hipFree(dev_springGraph));
	}
}

/// <summary>
/// Adjust the force acting on every particle based on the forces applied to its neighbors connected by springs
/// </summary>
template<int bloodCellCount, int particlesInBloodCell, int particlesStart, int springGraphStart>
__global__ static void gatherForcesKernel(BloodCells bloodCells)
{
	int indexInType = blockIdx.x * blockDim.x + threadIdx.x;
	if (indexInType >= particlesInBloodCell * bloodCellCount)
		return;

	int indexInCell = indexInType % particlesInBloodCell;
	int realIndex = particlesStart + indexInType;

	float3 position = bloodCells.particles.positions.get(realIndex);
	float3 velocity = bloodCells.particles.velocities.get(realIndex);
	float3 initialForce = bloodCells.particles.forces.get(realIndex);
	float3 newPosition {0,0,0}, newVelocity {0,0,0}, newForce{ 0, 0, 0 };

#pragma unroll
	for (int neighbourCellindex = 0; neighbourCellindex < particlesInBloodCell; neighbourCellindex++)
	{
		float springLength = bloodCells.dev_springGraph[springGraphStart + neighbourCellindex * particlesInBloodCell + indexInCell];

		if (springLength != NO_SPRING)
		{
			int neighbourIndex = realIndex - indexInCell + neighbourCellindex;

			float3 neighbourPosition = bloodCells.particles.positions.get(neighbourIndex);
			float3 neighbourVelocity = bloodCells.particles.velocities.get(neighbourIndex);
			float3 neighbourInitialForce = bloodCells.particles.forces.get(neighbourIndex);
			float3 p{ 0,0,0 }, v{ 0,0,0 };
			float3 springForceComponent = physics::calculateParticlesSpringForceComponent(position - neighbourPosition,
				velocity - neighbourVelocity, initialForce, neighbourInitialForce, springLength, p, v);

#ifdef USE_RUNGE_KUTTA_FOR_PARTICLE
			newPosition = newPosition + p;
			newVelocity = newVelocity + v;
#endif
			newForce = newForce + springForceComponent;
		}
	}
	// add gravitation and viscous damping
	newForce = newForce + physics::accumulateEnvironmentForcesForParticles(velocity);

#ifdef USE_RUNGE_KUTTA_FOR_PARTICLE
	//bloodCells.particles.positions.add(realIndex, newPosition);
	//bloodCells.particles.velocities.add(realIndex, newVelocity);
	bloodCells.particles.forces.set(realIndex, (initialForce + newForce)/6.0f);
#else
	bloodCells.particles.forces.set(realIndex, (initialForce + newForce) / 2);
#endif
}

void BloodCells::gatherForcesFromNeighbors(const std::array<hipStream_t, bloodCellTypeCount>& streams)
{
	using IndexList = mp_iota_c<bloodCellTypeCount>;
	mp_for_each<IndexList>([&](auto i)
		{
			using BloodCellDefinition = mp_at_c<BloodCellList, i>;
			constexpr int particlesStart = particleStarts[i];
			constexpr int graphStart = accumulatedGraphSizes[i];

			CudaThreads threads(BloodCellDefinition::count * BloodCellDefinition::particlesInCell);
			gatherForcesKernel<BloodCellDefinition::count, BloodCellDefinition::particlesInCell, particlesStart, graphStart>
				<< <threads.blocks, threads.threadsPerBlock, 0, streams[i] >> > (*this);
		});
}