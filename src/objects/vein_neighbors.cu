#include "vein_neighbors.cuh"

#include "../utilities/math.cuh"
#include "../utilities/cuda_handle_error.cuh"

#include <iostream>

VeinNeighbors::VeinNeighbors()
{
    const auto pair = calculateSpringLengths();
    const auto hostNeighbors = std::get<0>(pair);
    const auto hostSprings = std::get<1>(pair);

    int i = 0;
    for (auto&& pair : data)
    {   
        // Copy neighbor indices
        HANDLE_ERROR(hipMalloc((void**)&pair.ids, veinPositionCount * sizeof(int)));
        HANDLE_ERROR(hipMemcpy(pair.ids, hostNeighbors[i].data(), veinPositionCount * sizeof(int), hipMemcpyHostToDevice));

        // Copy spring lengths
        HANDLE_ERROR(hipMalloc((void**)&pair.springs, veinPositionCount * sizeof(float)));
        HANDLE_ERROR(hipMemcpy(pair.springs, hostSprings[i].data(), veinPositionCount * sizeof(float), hipMemcpyHostToDevice));
        i++;
    }
}

VeinNeighbors::VeinNeighbors(const VeinNeighbors& other) : isCopy(true)
{
    std::copy(other.data, other.data + veinVertexMaxNeighbors, data);
}

VeinNeighbors::~VeinNeighbors()
{
	if (isCopy)
        return;

    for (auto& [ids, springs] : data)
    {
        HANDLE_ERROR(hipFree(ids));
        HANDLE_ERROR(hipFree(springs));
    }
        
}
