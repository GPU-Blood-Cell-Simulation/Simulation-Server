#include "vein_neighbors.cuh"

#include "../utilities/math.cuh"
#include "../utilities/cuda_handle_error.cuh"

#include <iostream>

VeinNeighbors::VeinNeighbors(int gpuId) : isCopy(false)
{
    static const auto pair = calculateSpringLengths();
    static const auto hostNeighbors = std::get<0>(pair);
    static const auto hostSprings = std::get<1>(pair);

    int i = 0;
    CUDACHECK(hipSetDevice(gpuId));
    for (auto&& pair : data)
    {   
        // Copy neighbor indices
        CUDACHECK(hipMalloc((void**)&pair.ids, veinPositionCount * sizeof(int)));
        CUDACHECK(hipMemcpy(pair.ids, hostNeighbors[i].data(), veinPositionCount * sizeof(int), hipMemcpyHostToDevice));

        // Copy spring lengths
        CUDACHECK(hipMalloc((void**)&pair.springs, veinPositionCount * sizeof(float)));
        CUDACHECK(hipMemcpy(pair.springs, hostSprings[i].data(), veinPositionCount * sizeof(float), hipMemcpyHostToDevice));
        i++;
    }
    CUDACHECK(hipSetDevice(0));
}

VeinNeighbors::VeinNeighbors(const VeinNeighbors& other) : isCopy(true)
{
    std::copy(other.data, other.data + veinVertexMaxNeighbors, data);
}

VeinNeighbors& VeinNeighbors::operator=(const VeinNeighbors& other)
{
    isCopy = true;
    std::copy(other.data, other.data + veinVertexMaxNeighbors, data);
    return *this;
}

VeinNeighbors::~VeinNeighbors()
{
	if (isCopy)
        return;

    CUDACHECK(hipSetDevice(gpuId));
    for (auto&& [ids, springs] : data)
    {
        CUDACHECK(hipFree(ids));
        CUDACHECK(hipFree(springs));
    }
    CUDACHECK(hipSetDevice(0));
}
