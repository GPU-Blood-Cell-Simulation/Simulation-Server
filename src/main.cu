﻿#include <glad/glad.h>
#include "grids/uniform_grid.cuh"
#include "grids/no_grid.cuh"
#include "meta_factory/blood_cell_factory.hpp"
#include "meta_factory/vein_factory.hpp"
#include "objects/blood_cells.cuh"
#include "objects/vein_triangles.cuh"
#include "objects/cylindermesh.hpp"
#include "simulation/simulation_controller.cuh"
#include "utilities/cuda_handle_error.cuh"
#include "graphics/glcontroller.cuh"
#include "objects/cylindermesh.hpp"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream> // for debugging purposes
#include <sstream>
#include "hip/hip_runtime.h"
#include ""
#include "config/graphics.hpp"

#ifdef WINDOW_RENDER
#   include "graphics/windowcontroller.hpp"
#else

#   include "graphics/offscreencontroller.hpp"

#   undef __noinline__
#   include "graphics/streamingcontroller.hpp"
#   define __noinline__ __attribute__((noinline))

#endif

#define UNIFORM_TRIANGLES_GRID

//#pragma float_control( except, on )
//// NVIDIA GPU selector for devices with multiple GPUs (e.g. laptops)
//extern "C"
//{
//    __declspec(dllexport) unsigned long NvOptimusEnablement = 0x00000001;
//}

#ifdef WINDOW_RENDER
#define programLoopFunction void programLoop(WindowController& windowController)
#else
#define programLoopFunction void programLoop(StremmingController& streamingController)
#endif

programLoopFunction;

int main()
{
    // Choose which GPU to run on, change this on a multi-GPU system.
    HANDLE_ERROR(hipSetDevice(0));

#ifdef WINDOW_RENDER
    WindowController windowController;
#else
    OffscreeenController offscreenController;
    StremmingController streamingController("127.0.0.1", 4321);
    streamingController.StartStreaming();
#endif

    // Load GL and set the viewport to match window size
    gladLoadGL();
    glViewport(0, 0, windowWidth, windowHeight);

    glEnable(GL_DEPTH_TEST);
    glEnable(GL_CULL_FACE);
    VEIN_POLYGON_MODE = GL_FILL;

    // debug
    glEnable(GL_DEBUG_OUTPUT);

    // Main simulation loop
#ifdef WINDOW_RENDER
    programLoop(windowController);
#else
    programLoop(streamingController);
#endif

    // Cleanup
    HANDLE_ERROR(hipDeviceReset());

    return 0;
}

// Main simulation loop - upon returning from this function all memory-freeing destructors are called

programLoopFunction
{

    int frameCount = 0;

    // Create blood cells
    BloodCells bloodCells;

    // Create vein mesh
    // TODO: this will be unnecessary
    VeinGenerator veinGenerator;

    // Create vein triangles
    VeinTriangles triangles;
    SingleObjectMesh veinMesh = veinGenerator.CreateMesh();

    // Create grids
    UniformGrid particleGrid(particleCount, 20, 20, 20);
#ifdef UNIFORM_TRIANGLES_GRID
    UniformGrid triangleCentersGrid(triangleCount, 10, 10, 10);
#else
    NoGrid triangleCentersGrid;
#endif

    // Create the main simulation controller and inject its dependencies
    sim::SimulationController simulationController(bloodCells, triangles, &particleGrid, &triangleCentersGrid);
    
    // Create a graphics controller
    graphics::GLController glController(veinMesh, simulationController.initialCellPositions);
    graphics::Camera camera;
#ifdef WINDOW_RENDER
    double lastTime = glfwGetTime();
    windowController.ConfigureInputAndCamera(&camera);
#endif

    // MAIN LOOP HERE - dictated by glfw
    bool shouldBeRunning = true;
    while (shouldBeRunning)
    {
        // Clear 
        glClearColor(1.00f, 0.75f, 0.80f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

        // Calculate particle positions using CUDA
        simulationController.calculateNextFrame();
        // Pass positions to OpenGL
        glController.calculateTriangles(triangles);
        glController.calculatePositions(bloodCells.particles.positions);

        glController.draw(camera);

#ifdef WINDOW_RENDER // graphical render

        glfwSwapBuffers(windowController.window);

        // Show FPS in the title bar
        double currentTime = glfwGetTime();
        double delta = currentTime - lastTime;
        if (delta >= 1.0)
        {
            double fps = double(frameCount) / delta;
            std::stringstream ss;
            ss << "Blood Cell Simulation" << " " << " [" << fps << " FPS]";

            glfwSetWindowTitle(windowController.window, ss.str().c_str());
            lastTime = currentTime;
            frameCount = 0;
        }
        else
        {
            frameCount++;
        }

        // Handle user input
        glfwPollEvents();
        windowController.handleInput();

        shouldBeRunning = !glfwWindowShouldClose(windowController.window);
#else // server calculations

        // Send data to client
            // TODO
        streamingController.SendFrame();

        shouldBeRunning = frameCount++ < maxFrames;
#endif
    }
}